#include <hip/driver_types.h>
#include <torch/extension.h>

#include <iostream>
#include <vector>

#include "ATen/core/TensorBody.h"

using std::cerr, std::endl;

#include "basic_gemm.h"
#include "revisit_matmul.h"

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("simple_cutlass_gemm", &simple_cutlass_gemm, "gemm", py::arg("A"), py::arg("B"), py::arg("alpha") = 1.0F,
        py::arg("beta") = 0.0F);

  m.def("cutlass_half_gemm_relu", &cutlass_half_gemm_relu, "gemm", py::arg("A"), py::arg("B"), py::arg("bias"),
        py::arg("alpha") = 1.0F);

  m.def("matmul", &matmul, "matmul", py::arg("A"), py::arg("B"), py::arg("C") = py::none(), py::arg("version") = 0,
        py::arg("B_transposed") = false);
};
