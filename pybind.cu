#include "ATen/core/TensorBody.h"
#include <hip/driver_types.h>
#include <torch/extension.h>

#include <vector>
#include <iostream>

using std::cerr, std::endl;

#include "basic_gemm.h"

std::vector<torch::Tensor> virtually_anything(
  torch::Tensor &A,
  torch::Tensor &B
);

torch::Tensor simple_cutlass_gemm(
    torch::Tensor &A,
    torch::Tensor &B,
    float alpha,
    float beta) {
  int64_t M = A.size(0);
  int64_t N = B.size(1);
  int64_t K = A.size(1);
  auto options = A.options();
  torch::Tensor C = torch::zeros({M, N}, options);

  const float* A_ptr = A.data_ptr<float>();
  const float* B_ptr = B.data_ptr<float>();
  float* C_ptr = C.data_ptr<float>();

  int lda = int(K);
  int ldb = int(N);
  int ldc = int(N);

  hipError_t err = CutlassSgemmNN(
    (int)M, (int)N, (int)K,
    alpha,
    A_ptr, lda,
    B_ptr, ldb,
    beta,
    C_ptr, ldc
  );
  if (hipSuccess != err) {
    cerr << "Error in CutlassSgemmNN: " << hipGetErrorString(err) << endl;
  }
  return C;
}

torch::Tensor cutlass_half_gemm_relu(
  const torch::Tensor &A,
  const torch::Tensor &B,
  const torch::Tensor &bias,
  float alpha) {

  assert(A.scalar_type() == at::kHalf);
  assert(B.scalar_type() == at::kHalf);
  assert(bias.scalar_type() == at::kHalf);

  int M = A.size(0);
  int N = B.size(1);
  int K = A.size(1);
  auto options = A.options();
  torch::Tensor C = torch::zeros({M, N}, options);
  hipError_t err = CutlassHGemmRelu(
    M, N, K,
    alpha,
    static_cast<void *>(A.data_ptr()),
    K,
    static_cast<void *>(B.data_ptr()),
    N,
    static_cast<void *>(C.data_ptr()),
    N,
    static_cast<void *>(bias.data_ptr())
  );

  if (hipSuccess != err) {
    cerr << "Error in CutlassSgemmNN: " << hipGetErrorString(err) << endl;
  }

  return torch::zeros({1});
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("simple_cutlass_gemm", &simple_cutlass_gemm, "gemm",
  py::arg("A"),
  py::arg("B"),
  py::arg("alpha") = 1.0F,
  py::arg("beta") = 0.0F);

  m.def("cutlass_half_gemm_relu", &cutlass_half_gemm_relu, "gemm",
  py::arg("A"),
  py::arg("B"),
  py::arg("bias"),
  py::arg("alpha") = 1.0F);
};
