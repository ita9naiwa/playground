#include "hip/hip_runtime.h"
#include <iostream>

#include "ATen/core/TensorBody.h"
#include "ATen/ops/pad.h"
#include "c10/util/Exception.h"
#include "c10/util/Half.h"
#include "torch/nn/options/padding.h"
#include "wmma_matmul.h"

const int TILE_SIZE = 16;
using A_FRAGMENT = wmma::fragment<wmma::matrix_a, TILE_SIZE, TILE_SIZE, TILE_SIZE, half, wmma::row_major>;
using B_FRAGMENT = wmma::fragment<wmma::matrix_b, TILE_SIZE, TILE_SIZE, TILE_SIZE, half, wmma::row_major>;
using ACCM_FRAGMENT = wmma::fragment<wmma::accumulator, TILE_SIZE, TILE_SIZE, TILE_SIZE, half>;

__global__ void wmmaKernel(half *a, half *b, half *c, int M, int N, int K) {
  // Each warp loops i'th row of A and j'th column of B
  // to compute i'th row and j'th column of C.
  A_FRAGMENT a_frag;
  B_FRAGMENT b_frag;
  ACCM_FRAGMENT c_frag;

  wmma::fill_fragment(c_frag, __float2half(0.0f));

  half *a_tile = a + blockIdx.x * TILE_SIZE * K;
  half *b_tile = b + blockIdx.y * TILE_SIZE;
  for (int iter = 0; iter < (K / TILE_SIZE); ++iter) {
    wmma::load_matrix_sync(a_frag, a_tile + iter * TILE_SIZE, K);
    wmma::load_matrix_sync(b_frag, b_tile + iter * TILE_SIZE * N, N);
    wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
  }
  half *c_pos = c + blockIdx.x * TILE_SIZE * N + blockIdx.y * TILE_SIZE;
  wmma::store_matrix_sync(c_pos, c_frag, N, wmma::mem_row_major);
}

int next_multiple_of_16(int n) {
  int un = static_cast<unsigned int>(n);
  unsigned int ret = (un + 15U) & ~15U;
  return static_cast<int>(ret);
}

int ceil(int a, int b = TILE_SIZE) { return (a + b - 1) / b; }

torch::Tensor wmma_matmul(torch::Tensor A, torch::Tensor B, torch::Tensor C) {
  if (A.scalar_type() != torch::kHalf || B.scalar_type() != torch::kHalf || C.scalar_type() != torch::kHalf) {
    throw std::runtime_error("Input tensors must be of type torch::kHalf");
  }

  const int M = A.size(0);
  const int N = B.size(1);
  const int K = A.size(1);

  bool require_pad = false;
  if ((M % 16) || (N % 16) || (K % 16)) {
    TORCH_WARN(
        "Input dimensions are not multiples of 16."
        "Padding to the next multiple of 16.\n"
        "This will slow the compuation.");
    require_pad = true;
  }

  int _M = M;
  int _N = N;
  int _K = K;
  torch::Tensor A_padded = A;
  torch::Tensor B_padded = B;
  torch::Tensor C_padded = C;
  if (require_pad) {
    _M = next_multiple_of_16(M);
    _N = next_multiple_of_16(N);
    _K = next_multiple_of_16(K);

    // Note: Pad function in PyTorch interprets padding arguments in reverse order from the last dimension.
    torch::nn::functional::PadFuncOptions A_pad_options({0, _K - K, 0, _M - M});
    torch::nn::functional::PadFuncOptions B_pad_options({0, _N - N, 0, _K - K});
    torch::nn::functional::PadFuncOptions C_pad_options({0, _N - N, 0, _M - M});

    A_padded = torch::nn::functional::pad(A, A_pad_options);
    B_padded = torch::nn::functional::pad(B, B_pad_options);
    C_padded = torch::nn::functional::pad(C, C_pad_options);
  }

  dim3 grid(_M / TILE_SIZE, _N / TILE_SIZE);
  dim3 block(32);
  wmmaKernel<<<grid, block>>>((half *)A_padded.data_ptr<at::Half>(), (half *)B_padded.data_ptr<at::Half>(),
                              (half *)C_padded.data_ptr<at::Half>(), _M, _N, _K);
  return C_padded.index({torch::indexing::Slice(0, M), torch::indexing::Slice(0, N)});
}
