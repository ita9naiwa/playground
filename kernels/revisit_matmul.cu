#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include <cassert>
#include <iostream>

#include "ATen/Dispatch.h"
#include "ATen/ops/transpose.h"
#include "c10/core/ScalarType.h"
#include "revisit_matmul.h"
#include "wmma_matmul.h"

template <typename scalar_t>
__global__ void matmul_naive(scalar_t *A, scalar_t *B, scalar_t *C, int M, int K, int N, bool B_transposed) {
  int init_i = blockIdx.x * blockDim.x + threadIdx.x;
  int init_j = blockIdx.y * blockDim.y + threadIdx.y;

  int i_stride = gridDim.x * blockDim.x;
  int j_stride = gridDim.y * blockDim.y;

  for (int i = init_i; i < M; i += i_stride) {
    for (int j = init_j; j < N; j += j_stride) {
      scalar_t sum = 0;
      for (int k = 0; k < K; ++k) {
        if (!B_transposed)
          sum += A[i * K + k] * B[k * N + j];
        else
          sum += A[i * K + k] * B[j * K + k];
      }
      if (i < M && j < N) C[i * N + j] = sum;
    }
  }
}

template <typename scalar_t>
__global__ void matmul_smem(scalar_t *A, scalar_t *B, scalar_t *C, int M, int K, int N, bool B_transposed) {
  __shared__ scalar_t tile_A[16][16];
  __shared__ scalar_t tile_B[16][16];

  int init_i = blockIdx.x * blockDim.x;
  int init_j = blockIdx.y * blockDim.y;

  int i_stride = gridDim.x * blockDim.x;
  int j_stride = gridDim.y * blockDim.y;

  int i0 = threadIdx.x;
  int j0 = threadIdx.y;
  for (int i = init_i; i < M; i += i_stride) {
    for (int j = init_j; j < N; j += j_stride) {
      int sum = 0;
      for (int k = 0; k < K; k += 16) {  // Tiling over K dimension
        // Load A's tile into shared memory
        if (i + i0 < M && k + j0 < K)
          tile_A[i0][j0] = A[(i + i0) * K + (k + j0)];
        else
          tile_A[i0][j0] = 0;

        // Load B's tile into shared memory
        if (j + j0 < N && k + i0 < K)
          if (!B_transposed)
            tile_B[i0][j0] = B[(k + i0) * N + (j + j0)];
          else
            tile_B[i0][j0] = B[(j + j0) * N + (k + i0)];
        else
          tile_B[i0][j0] = 0;
        __syncthreads();

        // Perform computation within the tile
        for (int k0 = 0; k0 < 16; k0++) {  // Iterate over tile's K dimension
          sum += tile_A[i0][k0] * tile_B[k0][j0];
        }
        __syncthreads();
      }

      // Write the result to C
      if (i + i0 < M && j + j0 < N) {
        C[(i + i0) * N + (j + j0)] = sum;
      }
    }
  }
}

torch::Tensor matmul(torch::Tensor A, torch::Tensor B, std::optional<torch::Tensor> C, int version, bool B_transposed) {
  int M = A.size(0);
  int K = A.size(1);
  int N = B.size(1);

  assert(A.size(1) == B.size(0));

  torch::Tensor _C;
  if (!C.has_value())
    _C = torch::zeros({M, N}, A.options());
  else
    _C = C.value();

  if (version == 2) {
    return wmma_matmul(A, B, _C);
  }

  auto _B = B;
  if (B_transposed) _B = torch::transpose(B, 1, 0);

  dim3 block_size(16, 16);
  dim3 grid_size(64, 64);

  switch (version) {
    case 0:
      AT_DISPATCH_REDUCED_FLOATING_TYPES(A.scalar_type(), "matmul", [&] {
        matmul_naive<<<grid_size, block_size>>>(A.data_ptr<scalar_t>(), _B.data_ptr<scalar_t>(),
                                                _C.data_ptr<scalar_t>(), M, K, N, B_transposed);
      });
      break;
    case 1:
      AT_DISPATCH_REDUCED_FLOATING_TYPES(A.scalar_type(), "matmul", [&] {
        matmul_smem<<<grid_size, block_size>>>(A.data_ptr<scalar_t>(), B.data_ptr<scalar_t>(), _C.data_ptr<scalar_t>(),
                                               M, K, N, B_transposed);
      });
      break;
    default:
      throw std::runtime_error("Invalid version");
  }

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
  return _C;
}
